#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void add (int * a, int * b, int * c)
{
  *c = *a + *b;
}


int main (void)
{
  int a, b, c;
  int * d_a, * d_b, * d_c;
  int size = sizeof(int);
   
  // allocate space for device copies of a, b, c
  hipMalloc ((void**)&d_a, size);
  hipMalloc ((void**)&d_b, size);
  hipMalloc ((void**)&d_c, size);

  // setup input values
  a = 2;
  b = 7;
  
  // cuda input to device
  hipMemcpy (d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy (d_b, &b, size, hipMemcpyHostToDevice);

  // launch add() kernel on gpu
  add<<<1, 1>>>(d_a, d_b, d_c);

  // copy result back to host
  hipMemcpy (&c, d_c, size, hipMemcpyDeviceToHost);
  printf ("c = a + b = %d + %d = %d\n", a, b, c);

  // cleanup
  hipFree (d_a);
  hipFree (d_b);
  hipFree (d_c);

  return 0;
}
