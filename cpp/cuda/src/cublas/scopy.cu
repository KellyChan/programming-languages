/*
    cublasCopy - copy vector into vector

    - copy the vector x into the vector y
*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define n 6


int main(void)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float* x;
    float* y;
    x = (float*)malloc(n*sizeof(*x));
    for (j = 0; j < n; j++)
    {
        x[j] = (float)j;
    }
    printf("x: ");
    for (j = 0; j < n; j++)
    {
        printf("%2.0f,", x[j]);
    }
    printf("\n");
    y = (float*)malloc(n*sizeof(*y));

    // on the device
    float* d_x;
    float* d_y;
    cudaStat = hipMalloc((void**)&d_x, n*sizeof(*x));
    cudaStat = hipMalloc((void**)&d_y, n*sizeof(*y));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
    stat = hipblasScopy(handle, n, d_x, 1, d_y, 1);
    stat = hipblasGetVector(n, sizeof(float), d_y, 1, y, 1);
    
    printf("y after copy: \n");
    for (j = 0; j < n; j++)
    {
        printf("%2.0f,", y[j]);
    }
    printf("\n");

    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);

    free(x);
    free(y);
    
    return EXIT_SUCCESS;
  
}
