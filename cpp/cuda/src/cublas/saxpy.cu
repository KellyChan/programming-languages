/*

    cublasSaxpy: compute ax + y

        y = ax + y

*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define n 6


int main(void)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float* x;
    float* y;

    x = (float*)malloc (n*sizeof(*x));
    for (j = 0; j < n; j ++)
    {
        x[j] = (float)j;
    }
    
    y = (float*)malloc (n*sizeof(*y));
    for (j = 0; j < n; j++)
    {
        y[j] = (float)j;
    }
    printf("x, j:\n");

    for (j = 0; j < n; j++)
    {
        printf("%2.0f,", x[j]);
    }
    printf("\n");

    // on the device
    float* d_x;
    float* d_y;

    cudaStat = hipMalloc((void**)&d_x, n*sizeof(*x));
    cudaStat = hipMalloc((void**)&d_y, n*sizeof(*y));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
    stat = hipblasSetVector(n, sizeof(*y), y, 1, d_y, 1);

    float a1 = 2.0;
    stat = hipblasSaxpy(handle, n, &a1, d_x, 1, d_y, 1);
    stat = hipblasGetVector(n, sizeof(float), d_y, 1, y, 1);
    printf("y after Saxpy: \n");
    for (j = 0; j < n; j++)
    {
        printf("%2.0f,", y[j]);
    }
    printf("\n");
    
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
    free(x);
    free(y);
    
    return EXIT_SUCCESS;
}
