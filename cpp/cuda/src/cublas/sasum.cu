/*
    CUBLAS: sum of absolute values
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"

#define n 6  // length of x


int main(void)
{
    hipError_t cudaStat;  // cudaMalloc status
    hipblasStatus_t stat;  // CUBLAS functions status
    hipblasHandle_t handle; // CUBLAS context

    int j; // index of elements
    float* x;  // n-vector on the host
    x = (float*)malloc (n*sizeof(*x)); // host memory alloc
    for (j = 0; j < n; j++)
    {
        x[j] = (float)j;  // x = {0, 1, 2, 3, 4, 5}
    }
    printf("x: ");

    for (j = 0; j < n; j++)
    {
        printf("%2.0f,", x[j]); // print x
    }
    printf("\n");

    // on the device
    float* d_x;  // d_x - x on the device
    cudaStat = hipMalloc((void**)& d_x, n*sizeof(*x));
    stat = hipblasCreate(&handle);  // initialize CUBLAS context
    stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);; // cp x->d_x
    float result;

    // add absolute values of elements of the array d_x
    stat = hipblasSasum(handle, n, d_x, 1, &result);

    printf("sum of the absolute values of elements of x: %4.0f\n", result);

    hipFree(d_x);  // free device memory
    hipblasDestroy(handle); // destroy CUBLAS context
    free(x);
    return EXIT_SUCCESS;
}
