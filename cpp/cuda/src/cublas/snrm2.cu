/*
    cublasSnrm2 - Euclidean norm

    This function computes the Euclidean norm of the vector x
        |x| = sqrt(x0^2 + ... + x_(n-1)^2)

        where x = {x0,...,x_(n-1)}
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define n 6


int main(void)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float* x;
    x = (float*)malloc(n*sizeof(*x));
    for (j = 0; j < n; j++)
    {
        x[j] = (float)j;
    }
    printf("x:");
    for (j = 0; j < n; j++)
    {
        printf("%2.0f", x[j]);
    }
    printf("\n");

    // on the device
    float* d_x;
    cudaStat = hipMalloc((void**)& d_x, n*sizeof(*x));
    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
    float result;
    stat = hipblasSnrm2(handle, n, d_x, 1, &result);
    printf("Euclidean norm of x: ");
    printf("%7.3f\n", result);

    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    return EXIT_SUCCESS;
}
