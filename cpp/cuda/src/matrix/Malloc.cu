#include "hip/hip_runtime.h"
// Global variables
__constant__ float constData[256];
float data[256];
hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));


__device__ float devData;
float value = 3.14f;
hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));


__device__ float* devPointer;
float* ptr;
hipMalloc(&ptr, 256 * sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));


// Device Host: kernel definition
__global__ void 2DKernel(float* devPtr, size_t pitch, int width, int height)
{
    for (int r = 0; r < height; ++r)
    {
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c)
        {
            float element = row[c];
        }
    }
} 


__global__ void 3DKernel(hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
    char* devPtr = devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z)
    {
        char* slice = devPtr + z * slicePitch;
        for (int y = 0; y < height; ++y)
        {
            float* row = (float*)(slice + y * pitch);
            for (int x = 0; x < width; ++i)
            {
                float element = row[x];
            }
        }
    }
}


// Host code
int main()
{

    // Allocate memory with width x height
    int width = 64;
    int height = 64;
    float* devPtr;
    size_t pitch;
    hipMallocPitch(&devPtr, &pitch, width * sizeof(float), height);
    2DKernel<<<100, 512>>>(devPtr, pitch, width, height);

    // Allocate memory with width x height x depth
    int width = 64;
    int height = 64;
    int depth = 64;
    hipExtent extent = make_hipExtent(width * sizeof(float), height, depth));
    hipPitchedPtr devPitchedPtr;
    hipMalloc3D(&devPitchedPtr, extent);
    3DKernel<<<100, 512>>>(devPicthedPtr, width, height, depth);


}
