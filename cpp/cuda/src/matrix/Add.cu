
#include <hip/hip_runtime.h>
#include <stdio.h>

# define N 10


__global__ void AddKernel(int a, int b, int* c)
{
    *c = a + b;
}


__global__ void AddPlusKernel(int* a, int* b, int* c)
{
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}



int add(void)
{
    int c;
    int* d_c;
    hipMalloc((void**)&d_c, sizeof(int));
    AddKernel<<<1,1>>>(2, 7, d_c);
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("2 + 7 = %d\n", c);

    hipFree(d_c);

    return 0;
}


int addgpu(void)
{

    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    // Allocate the memory on the GPU
    hipMalloc((void**)&d_a, N*sizeof(int));
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&d_c, N*sizeof(int));

    // Fill the arrays 'a' and 'b' on the GPU
    for (int i=0; i<N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N*sizeof(int), hipMemcpyHostToDevice);

    AddPlusKernel<<<N, 1>>>(d_a, d_b, d_c);

    // Copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);

    // Display the results
    printf(" --- GPU --- \n");
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free the memory allocated on the GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;    
}


int main(void)
{
    add();
    addgpu();

    return 0;
}
