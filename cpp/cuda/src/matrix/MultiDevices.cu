
// Device Enumeration
int deviceCount;
CudaGetDeviceCount(&deviceCount);

int device;
for (device = 0; device < deviceCount; ++device)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d\n", device, deviceProp.major, deviceProp.minor);
}


// Device Selection
size_t size = 1024 * sizeof(float);

hipSetDevice(0);                        // set device 0 as current
float* p0;
hipMalloc(&p0, size);                   // allocate memory on device 0
MyKernel<<<1000, 128>>>(p0);             // launch kernel on device 0

hipSetDevice(1);                        // set device 1 as current
float* p1;
hipMalloc(&p1, size);                   // allocate memory on device 1
MyKernel<<<1000, 128>>>(p1);             // launch kernel ono device 1


// Stream and Event Behavior
hipSetDevice(0);
hipStream_t s0;
hipStreamCreate(&s0);
MyKernel<<<100, 64, 0, s0>>>();          // launch kernel on device 0 in s0

hipSetDevice(1);
hipStream_t s1;
hipStreamCreate(&s1);
MyKernel<<<100, 64, 0, s1>>>();          // launch kernel on device 1 in s1


// Peer-to-Peer Memory Access
size_t size = 1024 * sizeof(float);

hipSetDevice(0);
float* p0;
hipMalloc(&p0, size);

hipSetDevice(1);
float* p1;
hipMalloc(&p1, size);

hipSetDevice(0);
MyKernel<<<1000, 128>>>(p0);
hipSetDevice(1);
hipMemcpyPeer(p1, 1, p0, 0, size);     // Copy p0 to p1
MyKernel<<<1000, 128>>>(p1);            // launch kernel on device 1
