#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h> 


__global__ void init_stuff(hiprandState *state) {

 int idx = blockIdx.x * blockDim.x + threadIdx.x;

 hiprand_init(1337, idx, 0, &state[idx]);

}

__global__ void make_rand(hiprandState *state, float

*randArray) {

 int idx = blockIdx.x * blockDim.x + threadIdx.x;

 randArray[idx] = hiprand_uniform(&state[idx]);

}

void host_function() {

 hiprandState *d_state;

 hipMalloc(&d_state, nThreads * nBlocks);

 init_stuff<<<nblocks, nthreads>>>(d_state);

 make_rand<<<nblocks, nthreads>>>(d_state, randArray);

 hipFree(d_state);

}

host_function();
