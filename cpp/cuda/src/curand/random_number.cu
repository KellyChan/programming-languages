
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100


__global__ void random(int* result)
{
    // to keep track of the seed value
    // we will store a random state for every thread
    hiprandState_t state;

    // initialize the state
    // - 0: the seed controls the sequence of random values that are produced
    // - 0: the sequence number is only important with multiple core
    // - 0: the offset is how much extra we advance in the sequence for each call
    hiprand_init(0, 0, 0, &state);

    // curand works like rand - except that it takes a state as a parameter
    *result = hiprand(&state) % MAX;
}


int main()
{
    // allocate an int on the GPU
    int* gpu_x;
    hipMalloc((void**) &gpu_x, sizeof(int));

    // invoke the GPU to initialize all of the random states
    random<<<1,1>>>(gpu_x);

    // copy the random number back
    int x;
    hipMemcpy(&x, gpu_x, sizeof(int), hipMemcpyDeviceToHost);

    printf("Random number = %d.\n", x);

    // free the memory we allocated
    hipFree(gpu_x);

    return 0;
}
